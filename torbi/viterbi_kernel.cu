#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <chrono>

#include <vector>

#define NUM_THREADS 1024
#define WARP_SIZE 32
#define NUM_WARPS 32

#define FULL_MASK 0xffffffff

// Is this a good kernel? Maybe not. Does it work? Yes.
__global__ void viterbi_forward_kernel(
    float* __restrict__ observation, // BATCH x FRAMES x STATES
    int* __restrict__ batch_frames, // BATCH
    float* __restrict__ transition, // STATES x STATES
    float* __restrict__ initial, // STATES
    float* __restrict__ posterior, // BATCH x STATES
    int* __restrict__ memory, // BATCH x FRAMES x STATES
    int max_frames,
    int states
) {

    // Handle batch
    int batch_id = blockIdx.x;
    int frames = batch_frames[batch_id]; // Get number of frames for this batch item
    observation += batch_id * max_frames * states;
    posterior += batch_id * states;
    memory += batch_id * max_frames * states;

    // The id of the warp to which this thread belongs
    int warp_id = threadIdx.x / WARP_SIZE;
    // The id of this thread within its warp
    int thread_warp_id = threadIdx.x % WARP_SIZE;

    extern __shared__ float posterior_cache[];

    float *posterior_current = posterior_cache;
    float *posterior_next = posterior_cache+states;

    // Set initial
    for (int i=threadIdx.x; i<states; i+=NUM_THREADS) {
        posterior_current[i] = observation[i] + initial[i];
    }
    __syncthreads();

    for (int t=1; t<frames; t++) {
        // Get optimal
        // Iterate rows by warp (each warp gets assigned a row)
        int max_index;
        float max_value;
        for (int j=warp_id; j<states; j+=NUM_WARPS) {
            // __syncthreads();

            // Indices start out as just 0-WARP_SIZE for the first WARP_SIZE elements in the array
            max_index = thread_warp_id;
            // Values start as the first WARP_SIZE elements in the row, with row selected by j
            max_value = posterior_current[thread_warp_id] + transition[j*states+thread_warp_id];

            // Slide the warp over the row in a linear argmax search (parallelized by threads within the warp)
            // Note that we start here offset by the WARP_SIZE since we already initialized using the first chunk
            for (int i=thread_warp_id+WARP_SIZE; i<states; i+=WARP_SIZE) {
                // Get the new value from the current row at the current offset
                float new_value = posterior_current[i] + transition[j*states + i];
                if (new_value > max_value) {
                    max_index = i;
                    max_value = new_value;
                }
            }
            __syncwarp();

            // This is a first attempt at a parallel reduction
            for (int offset=WARP_SIZE/2; offset>0; offset/=2) {
                float new_value = __shfl_down_sync(FULL_MASK, max_value, offset);
                int new_index = __shfl_down_sync(FULL_MASK, max_index, offset);
                if (new_value > max_value) {
                    max_value = new_value;
                    max_index = new_index;
                }
            }
            if (thread_warp_id == 0) {
                memory[(t)*states+j] = max_index;
                // posterior[t*states+j] = observation[t*states+j] + max_value;
                posterior_next[j] = observation[t*states+j] + max_value;
            }
        }
        float *posterior_last = posterior_current;
        posterior_current = posterior_next;
        posterior_next = posterior_last; 
        __syncthreads();
    }

    // Write final posterior row
    for (int i=threadIdx.x; i<states; i+=NUM_THREADS) {
        posterior[i] = posterior_current[i];
    }
    __syncthreads();
}

void viterbi_forward_cuda(
    torch::Tensor observation,
    torch::Tensor batch_frames,
    torch::Tensor transition,
    torch::Tensor initial,
    torch::Tensor posterior,
    torch::Tensor memory,
    int max_frames,
    int states
) {
    const int threads = NUM_THREADS;

    int batch_size = observation.size(0);

    const dim3 blocks(batch_size);

    int device_num = observation.device().index();
    hipSetDevice(device_num);

    viterbi_forward_kernel<<<blocks, threads, 2*states*sizeof(float)>>>(
        observation.data<float>(),
        batch_frames.data<int>(),
        transition.data<float>(),
        initial.data<float>(),
        posterior.data<float>(),
        memory.data<int>(),
        max_frames,
        states
    );
}